#include "hip/hip_runtime.h"
#include "CUDAGoL.hpp"

#include <GL/gl.h>
#include <GL/glu.h>
#include <GL/glut.h>
#include <GL/freeglut_ext.h>
#include <ctime>

#define WHITE 1.0, 1.0, 1.0
#define BLACK 0.0, 0.0, 0.0


GLint FPS = 24;
GLint window_width = 600;
GLint window_height = 600;
GLfloat p_left = 0.0;
GLfloat p_right = 1.0;
GLfloat bottom = 0.0;
GLfloat top = 1.0;
GLint game_width = 100;
GLint game_height = 100;

CUDAGoL *game;

__device__ int countBottomLeft(
					int *grid,
					int width, 
					int height
					)
{

	int sum = 0;
	sum +=grid[1];
	sum +=grid[width]; //y=0
	sum +=grid[width+1];

	sum +=grid[width-1];
	sum +=grid[2*width-1];

	sum +=grid[(height-1)*width];
	sum +=grid[(height-1)*width+1];

	sum +=grid[height*width-1];

	return sum;
}

__device__ int countBottomRight(
					int *grid,
					int width,
					int height
					)
{

	int sum = 0;
	sum +=grid[width-2];
	sum +=grid[width*2-1]; //y=0
	sum +=grid[width*2 -2];

	sum +=grid[0];
	sum +=grid[width];

	sum +=grid[(height*width)-1];
	sum +=grid[(height*width)-2];

	sum += grid[(height-1)*width];

	return sum;	
}

__device__ int countTopLeft(
					int *grid,
					int width, 
					int height
				)
{

	int sum = 0;
	sum += grid[width*(height-1)+1];
	sum += grid[width*(height-2)]; //y=0
	sum += grid[width*(height-2)+1];

	sum += grid[(width*height)-1];
	sum += grid[(width*(height-1))-1];

	sum += grid[0];
	sum += grid[1];

	sum += grid[width-1];
	return sum;	

}

__device__ int countTopRight(
					int *grid,
					int width, 
					int height
				)
{

	int sum = 0;
	sum += grid[width-2];
	sum += grid[2*width-1];
	sum += grid[2*width-2];

	sum += grid[width*height-1];
	sum += grid[width*(height-1)-1];

	sum += grid[width-1];
	sum += grid[width-2];

	sum += grid[0];

	return sum;	
}

__device__ int countLeftEdge(
					int *grid,
					int width, 
					int height,
					int y
				)
{

	int sum = 0;
	sum += grid[(y+1)*width];
	sum += grid[(y+1)*width+1];
	sum += grid[y*width+1];
	sum += grid[(y-1)*width];
	sum += grid[(y-1)*width+1];

	sum += grid[(y+2)*width-1];
	sum += grid[(y+1)*width-1];
	sum += grid[y*width-1];

	return sum;	

}

__device__ int countRightEdge(
					int *grid,
					int width, 
					int height,
					int y
				)
{
	int sum = 0;
	sum += grid[(y+2)*width-1];
	sum += grid[(y+2)*width-2];
	sum += grid[(y+1)*width-2];
	sum += grid[y*width-2];
	sum += grid[y*width-1];

	sum += grid[(y+1)*width];
	sum += grid[y*width];
	sum += grid[(y-1)*width];

	return sum;	

}

__device__ int countBottomEdge(
					int *grid,
					int width, 
					int height,
					int x
				)
{
	int sum = 0;
	sum += grid[x-1];
	sum += grid[x+1];
	sum += grid[(width-1)+x-1];
	sum += grid[(width-1)+x];
	sum += grid[(width-1)+x+1];

	sum +=grid[(height -1)*width + x -1];
	sum +=grid[(height -1)*width + x];
	sum +=grid[(height -1)*width + x + 1];
	return sum;
}

__device__ int countTopEdge(
					int *grid,
					int width, 
					int height,
					int x
				)
{
	int sum = 0;
	sum += grid[((height-1)*width-1)+x-1];
	sum += grid[((height-1)*width-1)+x+1];
	sum += grid[((height-2)*width-1)+x-1];
	sum += grid[((height-2)*width-1)+x];
	sum += grid[((height-2)*width-1)+x+1];

	sum +=grid[x-1];
	sum +=grid[x];
	sum +=grid[x+1];
	return sum;
}

__device__ int countMiddle(
					int *grid,
					int width, 
					int height,
					int x,
					int y
				)
{

	int sum=0;
	int pos = y*width+x;
	sum +=grid[pos+1];
	sum +=grid[pos-1];
	pos = (y-1)*width+x;
	sum +=grid[pos];
	sum +=grid[pos-1];
	sum +=grid[pos+1];
	pos = (y+1)*width+x;
	sum +=grid[pos];
	sum +=grid[pos-1];
	sum +=grid[pos+1];

	return sum;

}
__device__ int countNeighbors(int *grid, int *width, int *height, int id){
	
	if(id==0) return countBottomLeft(grid, *width, *height);
	else if(id==*width-1) return countBottomRight(grid, *width, *height);
	else if(id==*width*(*height-1)) return countTopLeft(grid, *width, *height);
	else if(id==(*width**height)-1) return countTopRight(grid, *width, *height);
	else {

		int row = id/(*width);
		int col = id % *width;

		if( col == 0)
			return countLeftEdge(grid, *width, *height, row);
		
		else if( col == ((*width)-1))
			return countRightEdge(grid, *width, *height, row);
		
		else if(row == 0)
			return countBottomEdge(grid, *width, *height, col);

		else if(row == ((*height)-1))
			return countTopEdge(grid, *width, *height, col);

		else return countMiddle(grid, *width, *height, col, row);


	}
}


__device__ int execRules(int *grid, int *width, int *height, int id){
	
	int sum = countNeighbors(grid, width, height, id);

	if(grid[id]==0) return (sum ==3 || sum == 6)? 1 : 0;
	else return (sum == 2 || sum == 3)? 1 : 0;
}

__global__ void gol(int *grid, int *width, int *height)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    grid[id] = execRules(grid, width, height, id);
}


void closeprogram(unsigned char key, int x, int y){
	if(key==27)
		exit(0);
}

void display() {
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glLoadIdentity();
	
    GLfloat xSize = (p_right - p_left) / game_width;
	GLfloat ySize = (top - bottom) / game_height;
	
	glBegin(GL_QUADS);
	
	for (GLint x = 0; x < game_width; ++x) {
		for (GLint y = 0; y < game_height; ++y) {
            game->organismAt(x, y)? glColor3f(BLACK):glColor3f(WHITE);
            
			glVertex2f(    x*xSize+p_left,    y*ySize+bottom);
			glVertex2f((x+1)*xSize+p_left,    y*ySize+bottom);
			glVertex2f((x+1)*xSize+p_left,(y+1)*ySize+bottom);
			glVertex2f(    x*xSize+p_left,(y+1)*ySize+bottom);
		}
	}
	glEnd();
    	
	glFlush();
	glutSwapBuffers();
}


void reshape(int w, int h) {
	window_width = w;
	window_height = h;

	glViewport(0, 0, window_width, window_height);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(p_left, p_right, bottom, top);

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();

	glutPostRedisplay();
}

void iterate(){

	// Allocate memory for each vector on GPU
    hipMalloc(&(game->d_game), game->bytes);
    hipMalloc(&(game->d_width), sizeof(int));
    hipMalloc(&(game->d_height), sizeof(int));
    hipMemcpy( game->d_game, game->h_game, game->bytes, hipMemcpyHostToDevice);
    hipMemcpy( game->d_width, &game->width, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy( game->d_height, &game->height,sizeof(int), hipMemcpyHostToDevice);
    clock_t begin = clock();
 	gol<<<game->gridSize, game->blockSize>>>(game->d_game, game->d_width, game->d_height);
 	clock_t end = clock();
 	double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
 	fprintf(stderr, "%f\n", elapsed_secs);
    hipMemcpy( game->h_game, game->d_game, game->bytes, hipMemcpyDeviceToHost );
    hipFree(game->d_game);
    hipFree(game->d_width);
    hipFree(game->d_height);


}

void update(int value){
	
	iterate();

	glutPostRedisplay();
	glutTimerFunc(1000 / FPS, update, 0);
}

int main( int argc, char* argv[] )
{

  	glutInit(&argc, argv);
	
	glutInitWindowSize(window_width, window_height);
	glutInitWindowPosition(0, 0);
	glutCreateWindow("Game of Life");
	glClearColor(1, 1, 1, 1);

	glutKeyboardFunc(closeprogram);
	glutDisplayFunc(display);
	glutReshapeFunc(reshape);

	game = new CUDAGoL(window_width, window_height);
	update(0);
	glutMainLoop();
    return 0;
}